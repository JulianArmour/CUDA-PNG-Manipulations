#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "lodepng.h"

__global__ void rectify(unsigned char image[], int len) {
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  if (index > len) return;
  if (image[index] < 127)
    image[index] = 127;
}

int main(int argc, char** argv) {
  char* input_name = argv[1];
  char* output_name = argv[2];
  int no_threads = atoi(argv[3]);

  unsigned char *image, *d_image;
  unsigned int width, height;

  unsigned int error = lodepng_decode32_file((unsigned char**)&image, &width, &height, input_name);
  if (error) {
    printf("error %u: %s\n", error, lodepng_error_text(error));
    exit(-1);
  }

  hipMalloc((void**)&d_image, width * height * 4 * sizeof(unsigned char));
  hipMemcpy((void*)d_image, (void*)image, width * height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);

  //start timer
  float memsettime;
  hipEvent_t start, stop;
  hipEventCreate(&start); hipEventCreate(&stop);
  hipEventRecord(start, 0);

  rectify<<<(width * height * 4 + no_threads - 1) / no_threads, no_threads>>>(d_image, width * height * 4);

  //stop timer
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&memsettime, start, stop);
  printf("Kernel execution time: %f\n", memsettime);
  hipEventDestroy(start); hipEventDestroy(stop);

  hipMemcpy((void*)image, (void*)d_image, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

  lodepng_encode32_file(output_name, image, width, height);

  free(image);
  hipFree(d_image);

  return 0;
}
